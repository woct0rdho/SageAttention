#include "hip/hip_runtime.h"
#include "attn_cuda_sm89.h"
#include "qk_int_sv_f8_cuda_sm89.cuh"
torch::Tensor qk_int8_sv_f8_accum_f32_fuse_v_scale_attn(torch::Tensor query,
                    torch::Tensor key,
                    torch::Tensor value,
                    torch::Tensor output,
                    torch::Tensor query_scale,
                    torch::Tensor key_scale,
                    torch::Tensor value_scale,
                    int64_t tensor_layout,
                    int64_t is_causal,
                    int64_t qk_quant_gran,
                    double sm_scale,
                    int64_t return_lse)
{
  CHECK_CUDA(query);
  CHECK_CUDA(key);
  CHECK_CUDA(value);
  CHECK_CUDA(output);
  CHECK_CUDA(query_scale);
  CHECK_CUDA(key_scale);
  CHECK_CUDA(value_scale);

  CHECK_LASTDIM_CONTIGUOUS(query);
  CHECK_LASTDIM_CONTIGUOUS(key);
  CHECK_CONTIGUOUS(value); // ensure value is contiguous to prevent troubles in the kernel
  CHECK_LASTDIM_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(query_scale);
  CHECK_CONTIGUOUS(key_scale);
  CHECK_CONTIGUOUS(value_scale);

  CHECK_DTYPE(query, torch::kInt8);
  CHECK_DTYPE(key, torch::kInt8);
  // TODO: how to check fp8 data type?
  // CHECK_DTYPE(value, torch::kHalf);
  CHECK_DTYPE(query_scale, torch::kFloat32);
  CHECK_DTYPE(key_scale, torch::kFloat32);
  CHECK_DTYPE(value_scale, torch::kFloat32);

  CHECK_DIMS(query, 4);
  CHECK_DIMS(key, 4);
  CHECK_DIMS(value, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(query_scale, 3);
  CHECK_DIMS(key_scale, 3);
  CHECK_DIMS(value_scale, 3);

  const int batch_size = query.size(0);
  const int head_dim = query.size(3);

  int stride_bz_q = query.stride(0);
  int stride_bz_k = key.stride(0);
  int stride_bz_v = value.stride(0);
  int stride_bz_o = output.stride(0);

  int qo_len, kv_len, num_qo_heads, num_kv_heads;
  int stride_seq_q, stride_h_q, stride_seq_k, stride_h_k, stride_h_v, stride_d_v, stride_seq_o, stride_h_o;

  if (tensor_layout == 0)
  {
    qo_len = query.size(1);
    kv_len = key.size(1);
    num_qo_heads = query.size(2);
    num_kv_heads = key.size(2);

    stride_seq_q = query.stride(1);
    stride_h_q = query.stride(2);
    stride_seq_k = key.stride(1);
    stride_h_k = key.stride(2);
    stride_h_v = value.stride(2);
    stride_d_v = value.stride(1);
    stride_seq_o = output.stride(1);
    stride_h_o = output.stride(2);

    CHECK_SHAPE(key, batch_size, kv_len, num_kv_heads, head_dim);
    CHECK_SHAPE(output, batch_size, qo_len, num_qo_heads, head_dim);
    assert(value.size(1) == head_dim);
    assert(value.size(2) == num_kv_heads);
  }
  else
  {
    qo_len = query.size(2);
    kv_len = key.size(2);
    num_qo_heads = query.size(1);
    num_kv_heads = key.size(1);

    stride_seq_q = query.stride(2);
    stride_h_q = query.stride(1);
    stride_seq_k = key.stride(2);
    stride_h_k = key.stride(1);
    stride_h_v = value.stride(1);
    stride_d_v = value.stride(2);
    stride_seq_o = output.stride(2);
    stride_h_o = output.stride(1);

    CHECK_SHAPE(key, batch_size, num_kv_heads, kv_len, head_dim);
    CHECK_SHAPE(output, batch_size, num_qo_heads, qo_len, head_dim);
    assert(value.size(2) == head_dim);
    assert(value.size(1) == num_kv_heads);
  }

  if (num_qo_heads % num_kv_heads != 0) {
    std::ostringstream err_msg;
    err_msg << "num_qo_heads (" << num_qo_heads << ") must be divisible by num_kv_heads (" << num_kv_heads << ")";
    throw std::invalid_argument(err_msg.str());  
  }

  torch::Tensor lse = torch::empty({0});
  if (return_lse)
  {
    lse = torch::empty({batch_size, num_qo_heads, qo_len}, query.options().dtype(torch::kFloat32));
  }

  const int num_kv_groups = num_qo_heads / num_kv_heads;

  auto output_dtype = output.scalar_type();

  DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {
    DISPATCH_CAUSAL(is_causal, IS_CAUSAL, {
      DISPATCH_QK_QUANT_GRAN(qk_quant_gran, QK_QUANT_GRAN, {
        DISPATCH_RETURN_LSE(return_lse, RETURN_LSE, {  
          DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(output_dtype, DTypeOut, {
              
            constexpr int CTA_Q = 128;
            constexpr int CTA_K = 64;
            constexpr int WARP_Q = 32;
            constexpr int WARP_K = 64;

            assert(value.size(0) == batch_size);
            assert(value.size(3) >= div_ceil(kv_len, CTA_K) * CTA_K);

            constexpr MaskMode mask_mode = IS_CAUSAL ? MaskMode::kCausal : MaskMode::kNone;

            if constexpr (QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerWarp))
            {
              CHECK_SHAPE(query_scale, batch_size, num_qo_heads, div_ceil(qo_len, CTA_Q) * (CTA_Q / WARP_Q));
              CHECK_SHAPE(key_scale, batch_size, num_kv_heads, div_ceil(kv_len, CTA_K) * (CTA_K / WARP_K));
            }
            else if constexpr (QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerThread))
            {
              CHECK_SHAPE(query_scale, batch_size, num_qo_heads, div_ceil(qo_len, CTA_Q) * (CTA_Q / WARP_Q) * 8);
              CHECK_SHAPE(key_scale, batch_size, num_kv_heads, div_ceil(kv_len, CTA_K) * (CTA_K / WARP_K) * 4);    
            }
            else
            {
              static_assert(QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerWarp) || QK_QUANT_GRAN == static_cast<int>(QuantGranularity::kPerThread), "Unsupported quantization granularity");
            }

            CHECK_SHAPE(value_scale, batch_size, num_kv_heads, head_dim);

            //                                     smem_Q                                     smem_K                            smem_V                     smem_O
            size_t smem_max = std::max(CTA_Q * HEAD_DIM * sizeof(int8_t) + CTA_K * HEAD_DIM * sizeof(int8_t) + CTA_K * HEAD_DIM * sizeof(int8_t), CTA_Q * HEAD_DIM * sizeof(half));
            
            auto kernel_func = qk_int_sv_f8_attn_kernel<CTA_Q, CTA_K, WARP_Q, WARP_K, HEAD_DIM, DataType::kInt8, static_cast<QuantGranularity>(QK_QUANT_GRAN), static_cast<QuantGranularity>(QK_QUANT_GRAN),
                                                        float, false, DTypeOut, ComputeUnit::kCudaCore, mask_mode, RETURN_LSE, true, false, false>;

            hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max);

            dim3 grid(div_ceil(qo_len, CTA_Q), num_qo_heads, batch_size);
            dim3 block(32, (CTA_Q / WARP_Q) * (CTA_K / WARP_K));

            kernel_func<<<grid, block, smem_max>>>(
              query.data_ptr<int8_t>(), 
              key.data_ptr<int8_t>(),
              reinterpret_cast<int8_t*>(value.data_ptr()),
              reinterpret_cast<DTypeOut*>(output.data_ptr()),
              (RETURN_LSE) ? reinterpret_cast<float*>(lse.data_ptr()) : nullptr,
              reinterpret_cast<float*>(query_scale.data_ptr()),
              reinterpret_cast<float*>(key_scale.data_ptr()),
              reinterpret_cast<float*>(value_scale.data_ptr()),
              nullptr,
              qo_len,
              kv_len,
              num_kv_groups,
              stride_bz_q, stride_seq_q, stride_h_q,
              stride_bz_k, stride_seq_k, stride_h_k,
              stride_bz_v, stride_h_v, stride_d_v,
              stride_bz_o, stride_seq_o, stride_h_o,
              sm_scale);
          });
        });
      });
    });
  });

  return lse;
}